#include "hip/hip_runtime.h"
#include <cassert>
#include <stdexcept>

#include <crcham/codeword.hpp>
#include <crcham/crc.hpp>
#include <crcham/evaluator.hpp>
#include <crcham/math.hpp>

namespace crcham {

namespace {

template <class CRC>
__global__
void weightsKernel(size_t* weights, CRC crc, size_t message_bits, size_t error_bits) {
    // Allocate the minimum number of integers required to hold the message and FCS field
    size_t codeword_bits = message_bits + crc.length();
    size_t codeword_bytes = codeword_bits / 8;
    if (codeword_bits % 8 != 0) {
        codeword_bytes++;
    }
    auto codeword_byte_ptr = static_cast<uint8_t*>(
        malloc(codeword_bytes * sizeof(uint8_t)));

    const size_t widx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t pincr = gridDim.x * blockDim.x; 
    uint64_t pidx = blockIdx.x * blockDim.x + threadIdx.x; 
    uint64_t pmax = ncrll(codeword_bits, error_bits);
    size_t weight = 0;

    for (; pidx < pmax; pidx += pincr) {
        // Permute the bytes in the ${pidx}th way
        permute(codeword_byte_ptr, codeword_bytes, pidx, codeword_bits, error_bits);
        assert(popcount(codeword_byte_ptr, codeword_bytes) == error_bits); 
        // Test to see if the codeword with errors is considered valid
        uint64_t error_crc = extract(codeword_byte_ptr, codeword_bytes, codeword_bits, crc.length());
        uint64_t good_crc = crc.compute(codeword_byte_ptr, codeword_bytes);
        if (error_crc == good_crc) {
            weight++;
        }
    }
    weights[widx] = weight;

    free(codeword_byte_ptr);
}

}

WeightsEvaluator::WeightsEvaluator(uint64_t polynomial, size_t message_bits, size_t error_bits) 
    : d_polynomial(polynomial)
    , d_polylen(crcham::NaiveCRC(polynomial).length())
    , d_message(message_bits)
    , d_errors(error_bits)
    , d_evaluations(crcham::ncrll(message_bits + d_polylen, error_bits))
{
}

template<>
void WeightsEvaluator::run<true>()
{
    // Check that there is an available CUDA device
    {
        int devcnt = 0;
        hipGetDeviceCount(&devcnt);
        if (devcnt == 0) {
            throw std::runtime_error("A supported NVIDIA GPU could not be found.");
        }
    }

    // CPU should not busy-wait for the kernel to finish
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    // Find optimal block and grid sizes
    int grid_size;
    int block_size;
    hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, 
        crcham::weightsKernel<crcham::TabularCRC>);

    // Set maximum allowable memory sizes
    size_t original_heap;
    size_t required_heap = 2 * grid_size * block_size * (d_message / 8);
    hipDeviceGetLimit(&original_heap, hipLimitMallocHeapSize);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 
            std::max(original_heap, required_heap));

    // Allocate memory for thread-local weights
    size_t* weights;
    hipMallocManaged(&weights, grid_size * block_size * sizeof(size_t));
    hipMemset(weights, 0, grid_size * block_size * sizeof(size_t));

    // Run the kernel and block until it is done
    hipEvent_t start_event; 
    hipEvent_t stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event);
    if (d_polylen < 8) {
        crcham::NaiveCRC ncrc(d_polynomial);
        crcham::weightsKernel<crcham::NaiveCRC><<<grid_size, block_size>>>(
                weights, ncrc, d_message, d_errors); 
    }
    else {
        crcham::TabularCRC tcrc(d_polynomial);
        crcham::weightsKernel<crcham::TabularCRC><<<grid_size, block_size>>>(
                weights, tcrc, d_message, d_errors); 
    }
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    float millis = 0;
    hipEventElapsedTime(&millis, start_event, stop_event);
    d_elapsed = std::chrono::milliseconds((unsigned long) millis);

    // Accumulate results from all threads
    d_weight = 0;
    for (size_t i = 0; i < grid_size * block_size; i++) {
        d_weight += weights[i];
    }
    hipFree(weights);
}

template<>
void WeightsEvaluator::run<false>()
{
    throw std::runtime_error("Unimplemented.");
}

size_t WeightsEvaluator::evaluations() const {
    return d_evaluations;
}

size_t WeightsEvaluator::weight() const {
    return d_weight;
}

std::chrono::milliseconds WeightsEvaluator::elapsed() const {
    return d_elapsed;
}

}
