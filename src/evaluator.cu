#include "hip/hip_runtime.h"
#include <stdexcept>
#include <thread>

#include <crcham/codeword.hpp>
#include <crcham/crc.hpp>
#include <crcham/evaluator.hpp>
#include <crcham/math.hpp>

#include <omp.h>

namespace crcham {

namespace {

template <class CRC>
__global__
void weightsKernel(size_t* weights, CRC crc, size_t message_bits, size_t error_bits) {
    size_t codeword_bits = message_bits + crc.length();
    size_t codeword_bytes = codeword_bits / 8;
    if (codeword_bits % 8 != 0) {
        codeword_bytes++;
    }
    auto codeword = static_cast<uint8_t*>(
        malloc(codeword_bytes * sizeof(uint8_t)));

    const size_t widx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t pincr = gridDim.x * blockDim.x; 
    uint64_t pidx = blockIdx.x * blockDim.x + threadIdx.x; 
    uint64_t pmax = ncrll(codeword_bits, error_bits);
    size_t weight = 0;

    for (; pidx < pmax; pidx += pincr) {
        permute(codeword, codeword_bytes, pidx, codeword_bits, error_bits);
        uint64_t error_crc = extract(codeword, codeword_bytes, codeword_bits, crc.length());
        uint64_t good_crc = crc.compute(codeword, codeword_bytes);
        if (error_crc == good_crc) {
            weight++;
        }
    }
    weights[widx] = weight;

    free(codeword);
}

template <class CRC>
size_t weightsOpenMP(const CRC& crc, size_t message_bits, size_t error_bits) 
{
    size_t codeword_bits = message_bits + crc.length();
    size_t codeword_bytes = codeword_bits / 8;
    if (codeword_bits % 8 != 0) {
        codeword_bytes++;
    }

    auto num_threads = std::max(3u, std::thread::hardware_concurrency()) - 2;
    auto codewords = new uint8_t[num_threads * codeword_bytes]();
    auto weights = new size_t[num_threads]();
    uint64_t pmax = ncrll(codeword_bits, error_bits);

    #pragma omp parallel for num_threads(num_threads)
    for (uint64_t pidx = 0; pidx < pmax; pidx++) {
        auto codeword = codewords + codeword_bytes * omp_get_thread_num();
        permute(codeword, codeword_bytes, pidx, codeword_bits, error_bits);
        uint64_t error_crc = extract(codeword, codeword_bytes, codeword_bits, crc.length());
        uint64_t good_crc = crc.compute(codeword, codeword_bytes);
        if (error_crc == good_crc) {
            weights[omp_get_thread_num()]++;
        }
    }

    delete[] codewords;
    size_t weight = 0;
    for (size_t i = 0; i < num_threads; i++) {
        weight += weights[i];
    }
    return weight;
}

}

WeightsEvaluator::WeightsEvaluator(uint64_t polynomial, size_t message_bits, size_t error_bits) 
    : d_polynomial(polynomial)
    , d_polylen(crcham::NaiveCRC(polynomial).length())
    , d_message(message_bits)
    , d_errors(error_bits)
    , d_evaluations(crcham::ncrll(message_bits + d_polylen, error_bits))
{
}

template<>
void WeightsEvaluator::run<true>()
{
    // Check that there is an available CUDA device
    {
        int devcnt = 0;
        hipGetDeviceCount(&devcnt);
        if (devcnt == 0) {
            throw std::runtime_error("A supported NVIDIA GPU could not be found.");
        }
    }

    // CPU should not busy-wait for the kernel to finish
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    // Find optimal block and grid sizes
    int grid_size;
    int block_size;
    hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, 
        crcham::weightsKernel<crcham::TabularCRC>);

    // Set maximum allowable memory sizes
    size_t original_heap;
    size_t required_heap = 2 * grid_size * block_size * (d_message / 8);
    hipDeviceGetLimit(&original_heap, hipLimitMallocHeapSize);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 
            std::max(original_heap, required_heap));

    // Allocate memory for thread-local weights
    size_t* weights;
    hipMallocManaged(&weights, grid_size * block_size * sizeof(size_t));
    hipMemset(weights, 0, grid_size * block_size * sizeof(size_t));

    // Run the kernel and block until it is done
    hipEvent_t start_event; 
    hipEvent_t stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event);
    if (d_polylen < 8) {
        crcham::NaiveCRC ncrc(d_polynomial);
        crcham::weightsKernel<crcham::NaiveCRC><<<grid_size, block_size>>>(
                weights, ncrc, d_message, d_errors); 
    }
    else {
        crcham::TabularCRC tcrc(d_polynomial);
        crcham::weightsKernel<crcham::TabularCRC><<<grid_size, block_size>>>(
                weights, tcrc, d_message, d_errors); 
    }
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    float millis = 0;
    hipEventElapsedTime(&millis, start_event, stop_event);
    d_elapsed = std::chrono::milliseconds((unsigned long) millis);

    // Accumulate results from all threads
    d_weight = 0;
    for (size_t i = 0; i < grid_size * block_size; i++) {
        d_weight += weights[i];
    }
    hipFree(weights);
}

template<>
void WeightsEvaluator::run<false>()
{
    auto timestamp = std::chrono::steady_clock::now();
    if (d_polylen < 8) {
        crcham::NaiveCRC ncrc(d_polynomial);
        d_weight = weightsOpenMP(ncrc, d_message, d_errors);
    }
    else {
        crcham::TabularCRC tcrc(d_polynomial);
        d_weight = weightsOpenMP(tcrc, d_message, d_errors);
    }
    d_elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(
        std::chrono::steady_clock::now() - timestamp);
}

size_t WeightsEvaluator::evaluations() const {
    return d_evaluations;
}

size_t WeightsEvaluator::weight() const {
    return d_weight;
}

std::chrono::milliseconds WeightsEvaluator::elapsed() const {
    return d_elapsed;
}

}
