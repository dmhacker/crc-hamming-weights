#include "hip/hip_runtime.h"
#include <iostream>

#include <crcham/compute.hpp>
#include <crcham/codeword.hpp>
#include <crcham/crc.hpp>
#include <crcham/math.hpp>

namespace crcham {

namespace {

template <class CRC>
__global__
void hammingWeightKernel(size_t* weights, CRC crc, size_t message_bits, size_t error_bits) {
    // Allocate the minimum number of integers required to hold the message and FCS field
    size_t codeword_bits = message_bits + crc.length();
    size_t codeword_bytes = codeword_bits / 8;
    if (codeword_bits % 8 != 0) {
        codeword_bytes++;
    }
    auto codeword_byte_ptr = static_cast<uint8_t*>(
        malloc(codeword_bytes * sizeof(uint8_t)));

    const size_t widx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t pincr = gridDim.x * blockDim.x; 
    uint64_t pidx = blockIdx.x * blockDim.x + threadIdx.x; 
    uint64_t pmax = ncrll(codeword_bits, error_bits);
    size_t weight = 0;

    for (; pidx < pmax; pidx += pincr) {
        // Permute the bytes in the ${pidx}th way
        permute(codeword_byte_ptr, codeword_bytes, pidx, codeword_bits, error_bits);
        assert(popcount(codeword_byte_ptr, codeword_bytes) == error_bits); 
        // Test to see if the codeword with errors is considered valid
        uint64_t error_crc = extract(codeword_byte_ptr, codeword_bytes, codeword_bits, crc.length());
        uint64_t good_crc = crc.compute(codeword_byte_ptr, codeword_bytes);
        if (error_crc == good_crc) {
            weight++;
        }
    }
    weights[widx] = weight;

    free(codeword_byte_ptr);
}

}

size_t hammingWeightGPU(float* timing, uint64_t polynomial, size_t message_bits, size_t error_bits) 
{
    // Check that there is an available CUDA device
    {
        int devcnt = 0;
        hipGetDeviceCount(&devcnt);
        if (devcnt == 0) {
            throw std::runtime_error("A supported NVIDIA GPU could not be found.");
        }
    }

    // CPU should not busy-wait for the kernel to finish
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    // Find optimal block and grid sizes
    int grid_size;
    int block_size;
    hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, 
        crcham::hammingWeightKernel<crcham::TabularCRC>);

    // Set maximum allowable memory sizes
    size_t original_heap;
    size_t required_heap = 2 * grid_size * block_size * (message_bits / 8);
    hipDeviceGetLimit(&original_heap, hipLimitMallocHeapSize);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 
            std::max(original_heap, required_heap));

    // Allocate memory for thread-local weights
    size_t* weights;
    hipMallocManaged(&weights, grid_size * block_size * sizeof(size_t));
    hipMemset(weights, 0, grid_size * block_size * sizeof(size_t));

    // Run the kernel and block until it is done
    hipEvent_t start_event; 
    hipEvent_t stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event);
    size_t polylen = crcham::NaiveCRC(polynomial).length();
    if (polylen < 8) {
        crcham::NaiveCRC ncrc(polynomial);
        crcham::hammingWeightKernel<crcham::NaiveCRC><<<grid_size, block_size>>>(
                weights, ncrc, message_bits, error_bits); 
    }
    else {
        crcham::TabularCRC tcrc(polynomial);
        crcham::hammingWeightKernel<crcham::TabularCRC><<<grid_size, block_size>>>(
                weights, tcrc, message_bits, error_bits); 
    }
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(timing, start_event, stop_event);

    // Accumulate results from all threads
    size_t weight = 0;
    for (size_t i = 0; i < grid_size * block_size; i++) {
        weight += weights[i];
    }
    hipFree(weights);

    return weight;
}

size_t hammingWeightCPU(float* timing, uint64_t polynomial, size_t message_bits, size_t error_bits) 
{
    throw std::runtime_error("Unimplemented.");
}

}
