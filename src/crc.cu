#include "hip/hip_runtime.h"
#include <crcham/crc.hpp>

#include <cassert>

namespace crcham {

__device__ __host__
NaiveCRC::NaiveCRC(uint64_t koopman)
    : d_generator(koopman)
{
#ifdef __CUDA_ARCH__
    d_polylen = 64 - __clzll(koopman);
#else
    d_polylen = 64 - __builtin_clzll(koopman);
#endif
    if (d_polylen > 0) {
        d_generator ^= 1ULL << (d_polylen - 1);
    }
    d_generator <<= 1;
    d_generator |= 1;
}

__device__ __host__
uint64_t NaiveCRC::polynomial() const {
    return d_generator;
}

__device__ __host__
size_t NaiveCRC::length() const {
    return d_polylen;
}

__device__ __host__
uint64_t NaiveCRC::compute(const uint8_t* bytes, size_t bytelen) const {
    // NOTE: this is essentially a replication of a shift register
    uint64_t shiftr = 0;
    const size_t bitlen = bytelen * 8;
    for (size_t idx = 0; idx < bitlen + d_polylen; idx++) {
        auto bit = (idx < bitlen) ? 
            ((bytes[idx / 8] >> (7 - (idx % 8))) & 1) : 0;
        auto msb = (shiftr >> (d_polylen - 1)) & 1;
        shiftr <<= 1; 
        shiftr |= bit;
        shiftr &= (1ULL << d_polylen) - 1;
        shiftr ^= msb ? d_generator : 0;
    }
    return shiftr;
}

 __device__ __host__
TabularCRC::TabularCRC(uint64_t koopman) 
    : d_generator(koopman)
{
#ifdef __CUDA_ARCH__
    d_polylen = 64 - __clzll(koopman);
#else
    d_polylen = 64 - __builtin_clzll(koopman);
#endif
    assert(d_polylen >= 8);
    uint64_t mask = 1ULL << (d_polylen - 1);
    d_generator ^= mask;
    d_generator <<= 1;
    d_generator |= 1;
    for (uint64_t byte = 0; byte < 256; byte++) {
        uint64_t result = byte << (d_polylen - 8);
        for (size_t b = 0; b < 8; b++) {
            if (result & mask) {
                result <<= 1;
                result ^= d_generator;
            }
            else {
                result <<= 1;
            }
            result &= (1ULL << d_polylen) - 1;
        }
        d_table[byte] = result;
    }
}

__device__ __host__
uint64_t TabularCRC::polynomial() const {
    return d_generator;
}

__device__ __host__
size_t TabularCRC::length() const {
    return d_polylen;
}

__device__ __host__
uint64_t TabularCRC::compute(const uint8_t* bytes, size_t bytelen) const {
    // Compute_CRC32 at http://www.sunshine2k.de/articles/coding/crc/understanding_crc.html
    uint64_t crc = 0;
    for (size_t i = 0; i < bytelen; i++) {
        uint64_t msb = bytes[i];
        msb <<= (d_polylen - 8);
        msb ^= crc;
        size_t tidx = msb >> (d_polylen - 8);
        crc = (crc << 8) ^ d_table[tidx];
        crc &= (1ULL << d_polylen) - 1;
    }
    return crc;
}


}
