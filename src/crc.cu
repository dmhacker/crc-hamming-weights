#include "hip/hip_runtime.h"
#include <crcham/crc.hpp>

#include <cassert>

namespace crcham {

__device__ __host__
NaiveCRC::NaiveCRC(uint64_t koopman)
    : d_generator(koopman)
{
#ifdef __CUDA_ARCH__
    d_length = 64 - __clzll(koopman);
#else
    d_length = 64 - __builtin_clzll(koopman);
#endif
    if (d_length > 0) {
        d_generator ^= 1ULL << (d_length - 1);
    }
    d_generator <<= 1;
    d_generator |= 1;
}

__device__ __host__
uint64_t NaiveCRC::polynomial() const {
    return d_generator;
}

__device__ __host__
size_t NaiveCRC::length() const {
    return d_length;
}
 __device__ __host__
TabularCRC::TabularCRC(uint64_t koopman) 
    : d_generator(koopman)
{
#ifdef __CUDA_ARCH__
    d_length = 64 - __clzll(koopman);
#else
    d_length = 64 - __builtin_clzll(koopman);
#endif
    assert(d_length >= 8);
    uint64_t mask = 1ULL << (d_length - 1);
    d_generator ^= mask;
    d_generator <<= 1;
    d_generator |= 1;
    for (uint64_t byte = 0; byte < 256; byte++) {
        uint64_t result = byte << (d_length - 8);
        for (size_t b = 0; b < 8; b++) {
            if (result & mask) {
                result <<= 1;
                result ^= d_generator;
            }
            else {
                result <<= 1;
            }
            result &= (1ULL << d_length) - 1;
        }
        d_table[byte] = result;
    }
}

__device__ __host__
uint64_t TabularCRC::polynomial() const {
    return d_generator;
}

__device__ __host__
size_t TabularCRC::length() const {
    return d_length;
}


}
