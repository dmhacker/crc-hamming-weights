#include "hip/hip_runtime.h"
#include <crcham/fixed_width_integer.hpp>

#include <stdio.h>

namespace crcham {

namespace {
    __device__
    size_t ctz64(uint64_t x) 
    {
        // Assume x is not 0, otherwise this is undefined
        size_t result = 1;
        if ((x & 0xFFFFFFFF) == 0) {
            result += 32; 
            x = x >> 32;
        }
        if ((x & 0x0000FFFF) == 0) {
            result += 16; 
            x = x >>16;
        }
        if ((x & 0x000000FF) == 0) {
            result += 8; 
            x = x >> 8;
        }
        if ((x & 0x0000000F) == 0) {
            result += 4; 
            x = x >> 4;
        }
        if ((x & 0x00000003) == 0) {
            result += 2;
            x = x >> 2;
        }
        return result - (x & 1);
    }

    __device__
    uint64_t choose(uint64_t n, uint64_t k)
    {
        // See https://stackoverflow.com/questions/9330915/number-of-combinations-n-choose-r-in-c
        if (k > n) {
            return 0;
        }
        if (k * 2 > n) {
            k = n - k;
        }
        if (k == 0) {
            return 1;
        }
        uint64_t result = n;
        for( uint64_t i = 2; i <= k; ++i ) {
            result *= (n - i + 1);
            result /= i;
        }
        return result;
    }
}

__device__
FixedWidthInteger::FixedWidthInteger(FixedWidthBuffer& buffer)
    : d_buffer(buffer)
{
}

__device__
FixedWidthInteger&
    FixedWidthInteger::operator=(const FixedWidthInteger& fwint)
{
    // Copy fwint's buffer into our own.
    // NOTE: this assumes that our buffer is the same size as fwint's buffer
    memcpy(d_buffer.get(), fwint.d_buffer.get(), d_buffer.size() * sizeof(uint64_t));
    return *this;
}

__device__ 
size_t FixedWidthInteger::trailingZeroes()
{
    auto ptr = d_buffer.get();
    for (size_t i = 0; i < d_buffer.size() - 1; i++) {
        size_t j = d_buffer.size() - 1 - i;
        if (ptr[j] != 0) {
            return i * 64 + ctz64(ptr[j]);
        }
    }
    return (d_buffer.size() - 1) * 64 + umin(ctz64(ptr[0]), d_buffer.precision() % 64);
}

__device__ 
size_t FixedWidthInteger::hammingWeight() {
    auto ptr = d_buffer.get();
    size_t ones = 0;
    for (size_t i = 0; i < d_buffer.size(); i++) {
        ones += __popcll(ptr[i]);
    }
    return ones;
}

__device__ 
void FixedWidthInteger::operator|=(const FixedWidthInteger& other)
{
    auto ptr = d_buffer.get();
    auto optr = other.d_buffer.get();
    for (size_t i = 0; i < d_buffer.size(); i++) {
        ptr[i] |= optr[i];
    }
    ptr[0] &= d_buffer.leadingBitMask();
}

__device__ 
void FixedWidthInteger::operator&=(const FixedWidthInteger& other)
{
    auto ptr = d_buffer.get();
    auto optr = other.d_buffer.get();
    for (size_t i = 0; i < d_buffer.size(); i++) {
        ptr[i] &= optr[i];
    }
    ptr[0] &= d_buffer.leadingBitMask();
}

__device__ 
void FixedWidthInteger::operator>>=(size_t shifts)
{
    shifts = umin(shifts, d_buffer.precision());
    size_t element_shifts = shifts / 64; 
    size_t bit_shifts = shifts % 64;
    auto ptr = d_buffer.get();
    for (size_t i = 0; i < d_buffer.size() - element_shifts; i++) {
        size_t to = d_buffer.size() - i - 1;
        size_t from = to - element_shifts;
        uint64_t previous = from == 0 ? 0 : ptr[from - 1];
        ptr[to] = (ptr[from] >> bit_shifts) | (previous << (64 - bit_shifts));
    }
    for (size_t i = 0; i < element_shifts; i++) {
        ptr[i] = 0;
    }
    ptr[0] &= d_buffer.leadingBitMask();
}

__device__ 
void FixedWidthInteger::increment()
{
    auto ptr = d_buffer.get();
    for (size_t i = 0; i < d_buffer.size() - 1; i++) {
        size_t j = d_buffer.size() - 1 - i;
        if (ptr[j] + 1 == 0) {
            ptr[j] = 0;
        }
        else {
            ptr[j]++;
            return;
        }
    }
    if (ptr[0] == d_buffer.leadingBitMask()) {
        ptr[0] = 0;
    }
    else {
        ptr[0]++;
    }
}

__device__ 
void FixedWidthInteger::decrement()
{
    auto ptr = d_buffer.get();
    for (size_t i = 0; i < d_buffer.size() - 1; i++) {
        size_t j = d_buffer.size() - 1 - i;
        if (ptr[j] == 0) {
            ptr[j]--;
        }
        else {
            ptr[j]--;
            return;
        }
    }
    if (ptr[0] == 0) {
        ptr[0] = d_buffer.leadingBitMask();
    }
    else {
        ptr[0]--;
    }
}

__device__ 
void FixedWidthInteger::invert()
{
    auto ptr = d_buffer.get();
    for (size_t i = 0; i < d_buffer.size(); i++) {
        ptr[i] = ~ptr[i];
    }
    ptr[0] &= d_buffer.leadingBitMask();
}

__device__ 
void FixedWidthInteger::negate()
{
    invert();
    increment();
}

__device__ 
void FixedWidthInteger::permuteNext(FixedWidthInteger& tmp1, 
    FixedWidthInteger& tmp2) 
{
    auto& perm = *this;
    size_t ptz = trailingZeroes() + 1;
    tmp1 = perm;
    tmp1.decrement();
    perm |= tmp1;
    tmp1 = perm;
    tmp1.invert();
    tmp2 = tmp1;
    tmp2.negate();
    tmp1 &= tmp2;
    tmp1.decrement();
    tmp1 >>= ptz;
    perm.increment();
    perm |= tmp1;
}

__device__ 
void FixedWidthInteger::permuteNth(uint64_t n, size_t k) {
    size_t mmax = d_buffer.precision();
    for (size_t i = 0; i < mmax; i++) {
        size_t m = mmax - i;
        uint64_t m1ck = choose(m - 1, k); 
        uint64_t* ptr;
        if (i < mmax % 64) {
            ptr = d_buffer.get(); 
        }
        else {
            ptr = d_buffer.get() + (i - mmax % 64) / 64 + 1;
        }
        *ptr <<= 1;
        if (n >= m1ck) {
            *ptr |= 1;
            n -= m1ck;
            k--;
        }
    }

}

}
