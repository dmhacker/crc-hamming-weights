#include <crcham/cuda.hpp>

namespace crcham {

CUDA::CUDA() {
    int dcnt = 0;
    hipGetDeviceCount(&dcnt);
    hipDeviceProp_t device;
    for (int i = 0; i < dcnt; i++) {
        hipGetDeviceProperties(&device, i);
        d_devices.push_back(device);
    }
}

bool CUDA::enabled() const {
    return !d_devices.empty();
}

void CUDA::setup() const {
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
}

void CUDA::wait() const {
    hipDeviceSynchronize();
}

std::ostream & operator<<(std::ostream &os, const CUDA& cuda)
{
    return os << "Found CUDA device: " << cuda.d_devices[0].name << std::endl;
}

}
