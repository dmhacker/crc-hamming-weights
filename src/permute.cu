#include "hip/hip_runtime.h"
#include <crcham/math.hpp>
#include <crcham/permute.hpp>

namespace crcham {

__device__ __host__
void permute(uint32_t* arr, size_t len, uint64_t n, size_t m, size_t k) {
    memset(arr, 0, len * sizeof(uint32_t));
    for (size_t i = 0; i < m; i++) {
        uint64_t binom = ncrll(m - i - 1, k); 
        if (n >= binom) {
            arr[i / 32] |= (1 << (i % 32));
            n -= binom;
            k--;
        }
    }
}

__device__ __host__
size_t popcount(uint32_t* arr, size_t len) {
    size_t ones = 0;
    for (size_t i = 0; i < len; i++) {
#ifdef __CUDA_ARCH__
        ones += __popc(arr[i]);
#else
        ones += __builtin_popcount(arr[i]);
#endif
    }
    return ones;
}

}
