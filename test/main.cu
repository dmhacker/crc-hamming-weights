#include "hip/hip_runtime.h"
#include "all_tests.hpp"

#include <iostream>

__global__
void testKernel() {
    testFWBMetadata(17, 1, 131071);
    testFWBMetadata(64, 2, 0);
    testFWBMetadata(192, 4, 0);
    testFWBMetadata(201, 4, 511);
    testFWBMetadata(64000, 1001, 0);
    testFWBEquality(17);
    testFWBEquality(64);
    testFWBEquality(201);
    testFWBEquality(64000);
    testFWBInequality(17);
    testFWBInequality(64);
    testFWBInequality(201);
    testFWBInequality(64000);
    for (size_t p = 64; p <= 256; p += 3) {
        testFWIIncrement(p);
        testFWIDecrement(p);
        testFWIInvert(p);
        testFWIAnd(p);
        testFWIOr(p);
        for (size_t z = 0; z <= p; z++) {
            testFWITrailingZeroes(p, z);
        }
        for (size_t s = 0; s <= p * 2; s++) {
            testFWIRightShift(p, s);
        }
        // Chosen such that that ((p - 1) choose w) doesn't 
        // exceed the memory limits of a 64-bit unsigned integer
        for (size_t w = 1; w < 8; w++) {
            testFWIPermute(p, w);
        }
    }
}

int main() {
    int devices = 0;
    hipGetDeviceCount(&devices);
    if (devices == 0) {
        std::cerr << "Unable to find a CUDA-compatible GPU." << std::endl;
        return EXIT_FAILURE;
    }

    testKernel<<<1, 1>>>(); 
    std::cout << "Tests started." << std::endl;

    hipDeviceSynchronize();
    std::cout << "Tests finished." << std::endl;

    return EXIT_SUCCESS;
}
